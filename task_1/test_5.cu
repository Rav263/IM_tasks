
#include <hip/hip_runtime.h>
#include <stdio.h>
// Функция сложения двух векторов
__global__ void addVector(float* left, float* right, float* result)
{
  //Получаем id текущей нити.
  int idx = threadIdx.x;
  
  //Расчитываем результат.
  result[idx] = left[idx] + right[idx];
}


#define SIZE 512
__host__ int main()
{
  //Выделяем память под вектора
  float* vec1 = new float[SIZE];
  float* vec2 = new float[SIZE];
  float* vec3 = new float[SIZE];

  //Инициализируем значения векторов
  for (int i = 0; i < SIZE; i++)
  {
    vec1[i] = i;
    vec2[i] = i;
  }

  //Указатели на память видеокарте
  float* devVec1;
  float* devVec2;
  float* devVec3;

  //Выделяем память для векторов на видеокарте
  hipMalloc((void**)&devVec1, sizeof(float) * SIZE);
  hipMalloc((void**)&devVec2, sizeof(float) * SIZE);
  hipMalloc((void**)&devVec3, sizeof(float) * SIZE);

  //Копируем данные в память видеокарты
  hipMemcpy(devVec1, vec1, sizeof(float) * SIZE, hipMemcpyHostToDevice);
  hipMemcpy(devVec2, vec2, sizeof(float) * SIZE, hipMemcpyHostToDevice);


  dim3 gridSize = dim3(1, 1, 1);    //Размер используемого грида
  dim3 blockSize = dim3(SIZE, 1, 1); //Размер используемого блока

  //Выполняем вызов функции ядра
  addVector<<<gridSize, blockSize>>>(devVec1, devVec2, devVec3);
    
    //Выполняем вызов функции ядра
  //addVector<<<blocks, threads>>>(devVec1, devVec2, devVec3);

  //Хендл event'а
  hipEvent_t syncEvent;

  hipEventCreate(&syncEvent);    //Создаем event
  hipEventRecord(syncEvent, 0);  //Записываем event
  hipEventSynchronize(syncEvent);  //Синхронизируем event

  //Только теперь получаем результат расчета
  hipMemcpy(vec3, devVec3, sizeof(float) * SIZE, hipMemcpyDeviceToHost);

   //Результаты расчета
  for (int i = 0; i < SIZE; i++)
  {
    printf("Element #%i: %.1f\n", i , vec3[i]);
  }

  //
  // Высвобождаем ресурсы
  //

  hipEventDestroy(syncEvent);

  hipFree(devVec1);
  hipFree(devVec2);
  hipFree(devVec3);

  delete[] vec1; vec1 = 0;
  delete[] vec2; vec2 = 0;
  delete[] vec3; vec3 = 0;
}

